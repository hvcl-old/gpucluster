#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <omp.h>
int main (int argc, char *argv[])
{
	int id, np, i;
	char processor_name[MPI_MAX_PROCESSOR_NAME];
	int processor_name_len;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &np);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);
	MPI_Get_processor_name(processor_name, &processor_name_len);
	printf("Hello world from process %03d out of %03d, processor name %s\n",
		id, np, processor_name);
	int deviceCount = 0;
	
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(error_id));
        // exit(EXIT_FAILURE);
    }
    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
		#pragma omp parallel num_threads(deviceCount)
		{
			int d = omp_get_thread_num();
			#pragma omp critical
			{
				printf("This is GPU %d of processor name %s\n",
						d, processor_name);
			}
		}
    }

	MPI_Finalize();
	return 0;
}